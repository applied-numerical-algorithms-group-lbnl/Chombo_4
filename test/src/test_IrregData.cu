#include "EBProto.H"
#include <implem/Proto_IrregData.H>

void test_irreg_data_fill(double* ptr, std::vector<Proto::EBIndex<Proto::CELL>>& index, unsigned int size)
{
  for(int i = 0 ; i < size ; i++)
  {
    ptr[i] = i*2;
    Proto::Point p(i,0,0);
    Proto::EBIndex<Proto::CELL> e(p,i);
    index.push_back(e);
  }
}


bool test_irreg_data_check_fill(double* ptr, std::vector<Proto::EBIndex<Proto::CELL>>& index, unsigned int size)
{
  for(int i = 0 ; i < size ; i++)
  {
    if(ptr[i] != i*2) return false;
    Proto::Point p(i,0,0);
    Proto::EBIndex<Proto::CELL> e(p,i);
    if(index[i] !=e) return false;
  }
  return true;
}

bool test_irreg_data_check_set_zero(double* ptr, unsigned int size)
{
  for(int i = 0 ; i < size ; i++)
    if(ptr[i] != 0) return false;

  return true;
}

bool run_test_irreg_data_empty()
{
  Proto::IrregData<Proto::CELL,double,1> empty;

  bool check1 = !(empty.defined());
  bool check2 = empty.vecsize() == 0;
  bool check3 = empty.size() == 0;
  //bool check3 = !(empty.hasIndex(0));
  assert(check1);
  assert(check2);
  assert(check3);
  return check1 && check2 && check3;
}

bool run_test_irreg_data_has_index_empty()
{
  Proto::IrregData<Proto::CELL,double,1> empty;
  Proto::EBIndex<Proto::CELL> index;
  bool check = !(empty.hasIndex(index));
  assert(check);
  return check;
}

bool run_test_irreg_data_use_constructor()
{
  unsigned int size = 8;
  double* ptr = new double[size];
  std::vector<Proto::EBIndex<Proto::CELL>> index;
  Proto::Box bx(Proto::Point(0,0,0),Proto::Point(size-1,0,0));

  test_irreg_data_fill(ptr,index,size);

  bool sizeOK = size == index.size();
  if(!sizeOK) std::cout << " size: " << size << "!= index.size(): " << index.size() << std::endl;
  assert(size == index.size());

  bool before = test_irreg_data_check_fill(ptr,index,size);
  assert(before);

  // use this constructor to initialize data on the GPU
  Proto::IrregData<Proto::CELL,double,1> fill(bx, ptr, index);

  bool check = fill.size() == size;

#ifdef PROTO_CUDA
  double* checkPtr = new double[size];
  double* devicPtr = fill.data();
  protoMemcpy(checkPtr,devicPtr,size*sizeof(double),protoMemcpyDeviceToHost);
#else
  double* checkPtr = fill.data();
#endif  

  bool after = test_irreg_data_check_fill(checkPtr, *(fill.getIndicies()), size);
  assert(after);

  index.clear();
  free(ptr);  

  return check && before && after;
}

bool run_test_irreg_data_set_val()
{
  unsigned int size = 8;
  double* ptr = new double[size];
  std::vector<Proto::EBIndex<Proto::CELL>> index;
  Proto::Box bx(Proto::Point(0,0,0),Proto::Point(size-1,0,0));

  test_irreg_data_fill(ptr,index,size);
  Proto::IrregData<Proto::CELL,double,1> fill(bx, ptr, index);
  fill.setVal(0);

#ifdef PROTO_CUDA
  double* checkPtr = new double[size];
  double* devicPtr = fill.data();
  protoMemcpy(checkPtr,devicPtr,size*sizeof(double),protoMemcpyDeviceToHost);
#else
  double* checkPtr = fill.data();
#endif  

  // should be false
  bool nochange = test_irreg_data_check_fill(checkPtr, *(fill.getIndicies()), size);
  assert(!nochange);

  bool change = test_irreg_data_check_set_zero(checkPtr,size);

  index.clear();
  free(ptr);  

  return change && (!nochange);
}
