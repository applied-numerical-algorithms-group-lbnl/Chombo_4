#pragma once
#include "test_timer.H"
#include "test_IrregData.cu"

void run_perf_irreg_data_copy_args(unsigned int size)
{
  test_timer t_copy;

  double* ptr = new double[size];
  std::vector<Proto::EBIndex<Proto::CELL>> index;
  Proto::Box bx(Proto::Point(0,0,0),Proto::Point(size-1,0,0));

  test_irreg_data_fill(ptr,index,size);
  Proto::IrregData<Proto::CELL,double,1> in(bx, ptr, index);
  Proto::IrregData<Proto::CELL,double,1> out(bx, ptr, index);
  in.setVal(1);
  out.setVal(2);

  t_copy.begin();
  out.copy(in,bx,0,bx,0,1);
  t_copy.end();

  std::cout << " run_test_perf_copy_irreg_data_" << size << " ... " << t_copy.duration() << " ms or we copy" << size*1000/t_copy.duration() << " elem/s" << std::endl;

  index.clear();
  free(ptr);
}



void run_perf_irreg_linear_full_args(unsigned int size)
{
  test_timer t_linearIn, t_linearOut;
  double* ptr = new double[size];
  double* ptr2 = new double[size];
  double inNumber   = 1;
  double outNumber  = 2;
  std::vector<Proto::EBIndex<Proto::CELL>> index;
  Proto::Box bx(Proto::Point(0,0,0),Proto::Point(size-1,0,0));

  test_irreg_data_fill(ptr,index,size);
  index.clear();
  test_irreg_data_fill(ptr2,index,size);
  Proto::IrregData<Proto::CELL,double,1> in(bx, ptr2, index);
  Proto::IrregData<Proto::CELL,double,1> out(bx, ptr, index);

  void* inWork;
  size_t nBytes = in.charsize(bx,0,1);
  protoMalloc(DEVICE,inWork, nBytes);

  in.setVal(inNumber);
  out.setVal(outNumber);

  /* test copy */
  t_linearOut.begin();
  in.linearOut(inWork,bx,0,0); 
  t_linearOut.end();
  t_linearIn.begin();
  out.linearIn(inWork,bx,0,0); 
  t_linearIn.end();

  std::cout << " run_test_perf_irreg_data_linear_in_" << size << " ... " << t_linearIn.duration() << " ms" << std::endl;
  std::cout << " run_test_perf_irreg_data_linear_out_" << size << " ... " << t_linearOut.duration() << " ms" << std::endl;
  index.clear();
  free(ptr);  
  free(ptr2);  
  protoFree(DEVICE,inWork); 
}


void run_perf_irreg_linear_partial_args(unsigned int sizebox,unsigned int size)
{
  test_timer t_linearIn, t_linearOut;
  double* ptr = new double[size];
  double* ptr2 = new double[size];
  double inNumber   = 1;
  double outNumber  = 2;
  std::vector<Proto::EBIndex<Proto::CELL>> index;
  Proto::Box bx(Proto::Point(0,0,0),Proto::Point(size-1,0,0));

  unsigned int nbBoxes = size/sizebox;


  test_irreg_data_fill(ptr,index,size);
  index.clear();
  test_irreg_data_fill(ptr2,index,size);
  Proto::IrregData<Proto::CELL,double,1> in(bx, ptr2, index);
  Proto::IrregData<Proto::CELL,double,1> out(bx, ptr, index);

  void* inWork;

  unsigned int start = 0;
  unsigned int end = sizebox;
  size_t nBytes = 0;

  for(unsigned int i = 0; i < nbBoxes ; i++)
  {
    Proto::Box tmp(Proto::Point(start,0,0),Proto::Point(end-1,0,0));
    nBytes += in.charsize(tmp,0,1);
    start += sizebox;
    end += sizebox;
  }

  protoMalloc(DEVICE,inWork, nBytes);
  in.setVal(inNumber);
  out.setVal(outNumber);

  /* test copy */

  start = 0;
  end = sizebox;
  char* charWork = (char*)inWork;

  t_linearOut.begin();
  for(unsigned int i = 0; i < nbBoxes ; i++)
  {
    Proto::Box tmp(Proto::Point(start,0,0),Proto::Point(end-1,0,0));
    in.linearOut(charWork,tmp,0,0);
    charWork += in.charsize(tmp,0,1);
    start += sizebox;
    end += sizebox;
  }
  t_linearOut.end();

  start = 0;
  end = sizebox;
  charWork = (char*)inWork;

  t_linearIn.begin();
  for(unsigned int i = 0; i < nbBoxes ; i++)
  {
    Proto::Box tmp(Proto::Point(start,0,0),Proto::Point(end-1,0,0));
    out.linearIn(charWork,tmp,0,0);
    charWork += in.charsize(tmp,0,1);
    start += sizebox;
    end += sizebox;
  }
  t_linearIn.end();

  std::cout << " run_test_perf_irreg_data_linear_in_" << sizebox << "_" << size << " ... " << t_linearIn.duration() << " ms" << std::endl;
  std::cout << " run_test_perf_irreg_data_linear_out_" << sizebox << "_" << size << " ... " << t_linearOut.duration() << " ms" << std::endl;

  index.clear();
  free(ptr);
  free(ptr2);
  protoFree(DEVICE,inWork);
}


void run_perf_irreg_linear_partial_graph_args(unsigned int sizebox,unsigned int size)
{
  test_timer t_linearIn, t_linearOut;
  double* ptr = new double[size];
  double* ptr2 = new double[size];
  double inNumber   = 1;
  double outNumber  = 2;
  std::vector<Proto::EBIndex<Proto::CELL>> index;
  Proto::Box bx(Proto::Point(0,0,0),Proto::Point(size-1,0,0));

  unsigned int nbBoxes = size/sizebox;


  test_irreg_data_fill(ptr,index,size);
  index.clear();
  test_irreg_data_fill(ptr2,index,size);
  Proto::IrregData<Proto::CELL,double,1> in(bx, ptr2, index);
  Proto::IrregData<Proto::CELL,double,1> out(bx, ptr, index);

  void* inWork;

  unsigned int start = 0;
  unsigned int end = sizebox;
  size_t nBytes = 0;

  for(unsigned int i = 0; i < nbBoxes ; i++)
  {
    Proto::Box tmp(Proto::Point(start,0,0),Proto::Point(end-1,0,0));
    nBytes += in.charsize(tmp,0,1);
    start += sizebox;
    end += sizebox;
  }

  protoMalloc(DEVICE,inWork, nBytes);
  in.setVal(inNumber);
  out.setVal(outNumber);

  /* test copy */

  start = 0;
  end = sizebox;
  char* charWork = (char*)inWork;

  for(unsigned int i = 0; i < nbBoxes ; i++)
  {
    in.linearOut(charWork,bx,0,0);
  }

  start = 0;
  end = sizebox;
  charWork = (char*)inWork;

  hipStream_t stream;
  hipStreamCreate(&stream);
  hipGraph_t graph;
  hipGraphExec_t instance;

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  //{
    Proto::Box tmp(Proto::Point(start,0,0),Proto::Point(end-1,0,0));
  for(unsigned int i = 0; i < nbBoxes ; i++)
    out.linearIn(charWork,tmp,0,0);
  //  charWork += in.charsize(tmp,0,1);
    start += sizebox;
    end += sizebox;
  //}
  hipStreamEndCapture(stream, &graph);
  hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

  hipStreamSynchronize(stream);
  t_linearIn.begin();
  hipGraphLaunch(instance, stream);
  hipStreamSynchronize(stream);
  t_linearIn.end();

  std::cout << " run_test_perf_irreg_data_linear_in_" << sizebox << "_" << size << " ... " << t_linearIn.duration() << " ms" << std::endl;

  index.clear();
  free(ptr);
  free(ptr2);
  protoFree(DEVICE,inWork);
}
